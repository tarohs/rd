#include "hip/hip_runtime.h"
#include	"common.h"

#if defined(CUDA)	// while this module

#include	<hip/hip_runtime.h>
#if defined(CUDAMAP)
# include	"cvdraw.h"
#endif

float	*d_u, *d_v;
float	*d_lnx;
float	*d_flny;
float	*d_lsu, *d_lsv, *d_lu, *d_lv;
#if defined(CUDAMAP)
# define	ZMAP	(szxy * mag * mag * 3)
# define	ZZMAP	(ZMAP * sizeof(unsigned char))
unsigned char	*d_map;
#endif
#if defined(CUDAFMM)
float	*d_minmaxarr;
float	*d_minmaxres;	// {min(d_u), max(d_u), min(d_v), max(d_v)}
//Float	minmaxres[4];	// on host
#endif


__constant__ int	dc_szy, dc_szx, dc_szxy, dc_mag;
//__constant__ int	dc_pswitch, dc_edgetype;
__constant__ int	dc_edgetype;
__constant__ float	dc_du_dxdx, dc_dv_dxdx, dc_k;
__constant__ int	dc_pbase0, dc_pbase1;

#if defined(CUDALSOPT)
__global__ void	cudamklsk(float *d_u, float *d_v, float *d_lsu, float *d_lsv,
	int d_pswitch);
__global__ void	cudaupdatek(float *d_u, float *d_v, float *d_lu, float *d_lv,
	float *d_flny, float *d_lnx, int d_pswitch);
#elif defined(CUDABLOPT)
__global__ void cudagsblk(float *d_u, float *d_v, float *d_flny, float *d_lnx,
				int d_pswitch);
#else
__global__ void	cudagsk(float *d_u, float *d_v, float *d_flny, float *d_lnx,
				int d_pswitch);
#endif
#if defined(CUDAMAP)
void	cudamapbgr(void);
__global__ void cudamapk(float *d_u, float *d_v,
		unsigned char *d_map, int pswitch,
		float d_udif, float d_umin, float d_vdif, float d_vmin);
#endif
#if defined(CUDAFMM)
void	cudafindminmax(void);
__global__ void cudafindminmaxk(float *source, int size, float *result);
#endif


void
cudamallocmx(void)
{
	if (hipMalloc((void **)&d_u, UVMATRIXSIZE)
			!= hipSuccess ||
		hipMalloc((void **)&d_v, UVMATRIXSIZE)
			!= hipSuccess ||
		hipMalloc((void **)&d_lnx, sizeof(float) * szx)
			!= hipSuccess ||
		hipMalloc((void **)&d_flny, sizeof(float) * szy)
			!= hipSuccess
	   ) {
		fprintf(stderr, "(cudamallocmx) %s\n",
			hipGetErrorString(hipGetLastError()));
		exit(31);
	}
//	printf("u %p, v %p +%x\n", d_u, d_v, UVMATRIXSIZE);
#if defined(CUDALSOPT)
	if (hipMalloc((void **)&d_lsu, sizeof(float) * szxy)
			!= hipSuccess ||
		hipMalloc((void **)&d_lsv, sizeof(float) * szxy)
			!= hipSuccess
		) {
		exit(32);
	}
#endif
#if defined(CUDAMAP)
	if (hipMalloc((void **)&d_map, ZZMAP) != hipSuccess) {
		exit(32);
	}
#endif
#if defined(CUDAFMM)
	int		nsrch = (szxy + CUDA_FMMSIZE - 1) / CUDA_FMMSIZE;
	if (hipMalloc((void **)&d_minmaxarr, sizeof(float) * 2 * nsrch)
		 != hipSuccess ||
		hipMalloc((void **)&d_minmaxres, sizeof(float) * 4)
		 != hipSuccess
	   ) {
		exit(32);
	}
#endif

	return;
}


void
cudafreemx(void)
{
	hipFree(&d_u);
	hipFree(&d_v);
	hipFree(&d_lnx);
	hipFree(&d_flny);
#if defined(CUDALSOPT)
	hipFree(&d_lsu);
	hipFree(&d_lsv);
#endif
#if defined(CUDAMAP)
	hipFree(&d_map);
#endif

	return;
}



void
cudaupload(void)
{
	float	du_dxdx, dv_dxdx;

	du_dxdx = Currentparamset.du / dx / dx;
	dv_dxdx = Currentparamset.dv / dx / dx;
	if (hipMemcpyToSymbol(HIP_SYMBOL(dc_du_dxdx), &du_dxdx, sizeof(float))
			!= hipSuccess ||
		hipMemcpyToSymbol(HIP_SYMBOL(dc_dv_dxdx), &dv_dxdx, sizeof(float))
			!= hipSuccess ||
		hipMemcpyToSymbol(HIP_SYMBOL(dc_k),  &Currentparamset.k,  sizeof(float))
			!= hipSuccess ||
	    hipMemcpyToSymbol(HIP_SYMBOL(dc_szy), &szy, sizeof(int))
			!= hipSuccess ||
		hipMemcpyToSymbol(HIP_SYMBOL(dc_szx), &szx, sizeof(int))
			!= hipSuccess ||
		hipMemcpyToSymbol(HIP_SYMBOL(dc_szxy), &szxy, sizeof(int))
			!= hipSuccess ||
		hipMemcpyToSymbol(HIP_SYMBOL(dc_mag), &mag, sizeof(int))
			!= hipSuccess ||
		hipMemcpyToSymbol(HIP_SYMBOL(dc_pbase0), &pbase0, sizeof(int))
			!= hipSuccess ||
		hipMemcpyToSymbol(HIP_SYMBOL(dc_pbase1), &pbase1, sizeof(int))
			!= hipSuccess ||
		hipMemcpyToSymbol(HIP_SYMBOL(dc_edgetype), &edgetype, sizeof(int))
			!= hipSuccess
		) {
		exit(33);
	}
// half sized copy don't work... why?
//	if (hipMemcpy(d_u + pswitch * zszxy, umatrix + pswitch * zszxy, zszxy,
//			hipMemcpyHostToDevice) != hipSuccess ||
//		hipMemcpy(d_v + pswitch * zszxy, vmatrix + pswitch * zszxy, zszxy,
//			hipMemcpyHostToDevice) != hipSuccess ||
	if (hipMemcpy(d_u, umatrix, UVMATRIXSIZE,
			hipMemcpyHostToDevice) != hipSuccess ||
		hipMemcpy(d_v, vmatrix, UVMATRIXSIZE,
			hipMemcpyHostToDevice) != hipSuccess
	   ) {
		exit(34);
	}
	return;
}


void
cudadownload(void)
{
//	if (hipMemcpy(umatrix + pswitch * zszxy, d_u + pswitch * zszxy, zszxy,
//			hipMemcpyDeviceToHost) != hipSuccess ||
//		hipMemcpy(vmatrix + pswitch * zszxy, d_v + pswitch * zszxy, zszxy,
//			hipMemcpyDeviceToHost) != hipSuccess
	if (hipMemcpy(umatrix, d_u, UVMATRIXSIZE,
			hipMemcpyDeviceToHost) != hipSuccess ||
		hipMemcpy(vmatrix, d_v, UVMATRIXSIZE,
			hipMemcpyDeviceToHost) != hipSuccess
	   ) {
		exit(35);
	}
	return;
}


void
cudachangelnx(void)
{
	if (hipMemcpy(d_lnx, lnx, sizeof(float) * szx,
			hipMemcpyHostToDevice) != hipSuccess) {
		exit(36);
	}
	return;
}


void
cudachangeflny(void)
{
	if (hipMemcpy(d_flny, flny, sizeof(float) * szy,
			hipMemcpyHostToDevice) != hipSuccess) {
		exit(37);
	}
	return;
}


void
cudaiter(void)
{
	dim3	dimBlock(CUDA_BLOCKSIZE / 2, CUDA_BLOCKSIZE);
#if defined(CUDABLOPT)
	dim3	dimGrid((szx + CUDA_BLOCKSIZE - 2 - 1) / (CUDA_BLOCKSIZE - 2),
			        (szy + CUDA_BLOCKSIZE - 2 - 1) / (CUDA_BLOCKSIZE - 2));
#else
	dim3	dimGrid((szx + CUDA_BLOCKSIZE - 1) / CUDA_BLOCKSIZE,
			        (szy + CUDA_BLOCKSIZE - 1) / CUDA_BLOCKSIZE);
#endif
	hipError_t		stat;

	if (hipMemcpyToSymbol(HIP_SYMBOL(dc_pbase0), &pbase0, sizeof(int))
			!= hipSuccess ||
		hipMemcpyToSymbol(HIP_SYMBOL(dc_pbase1), &pbase1, sizeof(int))
			!= hipSuccess
	   ) {
		exit(38);
	}
	for (Int g = 0; g < anistop; g++) {
#if defined(CUDALSOPT)
		cudamklsk<<<dimGrid, dimBlock>>>(d_u, d_v, d_lsu, d_lsv, pswitch);
		if ((stat = hipGetLastError()) != hipSuccess) {
			fprintf(stderr, "(kernel) %s\n", hipGetErrorString(stat));
			exit(39);
		}
		cudaupdatek<<<dimGrid, dimBlock>>>(d_u, d_v, d_lsu, d_lsv,
			d_flny, d_lnx,
			pswitch);
		if ((stat = hipGetLastError()) != hipSuccess) {
			fprintf(stderr, "(kernel) %s\n", hipGetErrorString(stat));
			exit(40);
		}
#elif defined(CUDABLOPT)
		cudagsblk<<<dimGrid, dimBlock>>>(d_u, d_v, d_flny, d_lnx,
			pswitch);
		if ((stat = hipGetLastError()) != hipSuccess) {
			fprintf(stderr, "(kernel) %s\n", hipGetErrorString(stat));
			exit(41);
		}
#else
		cudagsk<<<dimGrid, dimBlock>>>(d_u, d_v, d_flny, d_lnx,
			pswitch);
		if ((stat = hipGetLastError()) != hipSuccess) {
			fprintf(stderr, "(kernel) %s\n", hipGetErrorString(stat));
			exit(42);
		}
#endif
		Switchplain();
	}

	return;
}


//
// CUDA kernel functions
//


#if ! defined(CUDALSOPT) && ! defined(CUDABLOPT)	// simple method
__global__ void
cudagsk(float *d_u, float *d_v, float *d_flny, float *d_lnx, int d_pswitch)
{
	int		gy = blockIdx.y * blockDim.y + threadIdx.y;
    int		gx = blockIdx.x * blockDim.x + threadIdx.x;
	int		x, y, pnow, plf, prt, pab, pbl;
	float	lu, lv;
	float	uvv;
	int		d_pbasenow = (d_pswitch ? dc_pbase1 : dc_pbase0);
	int		d_pbasenxt = (d_pswitch ? dc_pbase0 : dc_pbase1);

	if (dc_szy <= gy || dc_szx <= gx) {	// for block padding threads
		return;
	}
#if defined(CUDAINTERLEAVE)	// swap bit (0, 1) and (2, 3)
	gy = (gy & ~0xf) | ((gy & 0xc) >> 2) | ((gy & 0x3) << 2);
	gx = (gx & ~0xf) | ((gx & 0xc) >> 2) | ((gx & 0x3) << 2);
#endif
	if (dc_edgetype == EDGE_NEUMANN) {
		if (gy == 0) {
			y = 1;
		} else if (gy == dc_szy - 1) {
			y = dc_szy - 2;
		} else {
			y = gy;
		}
		if (gx == 0) {
			x = 1;
		} else if (gx == dc_szx - 1) {
			x = dc_szx - 2;
		} else {
			x = gx;
		}
	} else {	// EDGE_DONUT
		y = gy;
		x = gx;
	}
#define	D_f	(d_flny[y] * d_lnx[x])

	pnow = d_pbasenow + y * dc_szx + x;
	pab = pnow - dc_szx;
	pbl = pnow + dc_szx;
	if (y == 0) {
		pab = pab + dc_szxy;
	} else if (y == dc_szy - 1) {
		pbl = pbl - dc_szxy;
	}
	plf = pnow - 1;
	prt = pnow + 1;
	if (x == 0 && y == 0) {
		plf = plf + dc_szxy;
	} else if (x == dc_szx - 1 && y == dc_szy - 1) {
		prt = prt - dc_szxy;
	}

#define		Pnxt 	(d_pbasenxt + gy * dc_szx + gx)
#define		D_unow	d_u[pnow]
#define		D_vnow	d_v[pnow]
	lu = (
			d_u[pab]		// Unow(y - 1, x)
			+ d_u[plf]		// Unow(y, x - 1)
			+ d_u[prt]		// Unow(y, x + 1)
			+ d_u[pbl]		// Unow(y + 1, x)
			- 4. * D_unow	// Unow(y, x)
		) * dc_du_dxdx;
	lv = (
			d_v[pab]		// Vnow(y - 1, x)
			+ d_v[plf]		// Vnow(y, x - 1)
			+ d_v[prt]		// Vnow(y, x + 1)
			+ d_v[pbl]		// Vnow(y + 1, x)
			- 4. * D_vnow	// Vnow(y, x)
		) * dc_dv_dxdx;
	uvv	= D_unow * D_vnow * D_vnow;
	d_u[Pnxt]					// Unxt(gy, gx) 
		= D_unow + DT * (lu + -uvv + D_f * (1. - D_unow));
	d_v[Pnxt]					// Vnxt(gy, gx)
		= D_vnow + DT * (lv + uvv - (D_f + dc_k) * D_vnow);

	return;
}
#endif	// simple method


#if defined(CUDALSOPT)	// laprasian optimized

__global__ void
cudamklsk(float *d_u, float *d_v, float *d_lsu, float *d_lsv, int d_pswitch)
{
	int		y = blockIdx.y * blockDim.y + threadIdx.y;
    int		x = blockIdx.x * blockDim.x + threadIdx.x;
	int		ppnow, pplf, ppab;
	int		d_pbasenow = (d_pswitch ? dc_pbase1 : dc_pbase0);
//	int		d_pbasenxt = (d_pswitch ? dc_pbase0 : dc_pbase1);

	if (dc_szy <= y || dc_szx <= x) {	// for block padding threads
		return;
	}
	ppnow = y * dc_szx + x;
	pplf = ppnow - 1;
	ppab = ppnow - dc_szx;
	if (pplf < 0) {
		pplf = pplf + dc_szxy;
	}
	if (ppab < 0) {
		ppab = ppab + dc_szxy;
	}
	d_lsu[ppnow] = d_u[d_pbasenow + pplf] + d_u[d_pbasenow + ppab];
	d_lsv[ppnow] = d_v[d_pbasenow + pplf] + d_v[d_pbasenow + ppab];
	
	return;
}



__global__ void
cudaupdatek(float *d_u, float *d_v, float *d_lsu, float *d_lsv,
	float *d_flny, float *d_lnx, int d_pswitch)
{
	int		gy = blockIdx.y * blockDim.y + threadIdx.y;
    int		gx = blockIdx.x * blockDim.x + threadIdx.x;
	int		x, y, pnow, ppnow, pprb;
	int		d_pbasenow = (d_pswitch ? dc_pbase1 : dc_pbase0);
	int		d_pbasenxt = (d_pswitch ? dc_pbase0 : dc_pbase1);
	float	uvv;

	if (dc_szy <= gy || dc_szx <= gx) {	// for block padding threads
		return;
	}
	if (dc_edgetype == EDGE_NEUMANN) {
		if (gy == 0) {
			y = 1;
		} else if (gy == dc_szy - 1) {
			y = dc_szy - 2;
		} else {
			y = gy;
		}
		if (gx == 0) {
			x = 1;
		} else if (gx == dc_szx - 1) {
			x = dc_szx - 2;
		} else {
			x = gx;
		}
	} else {	// EDGE_DONUT
		y = gy;
		x = gx;
	}
#define	D_f	(d_flny[y] * d_lnx[x])

	ppnow = y * dc_szx + x;
	pnow = d_pbasenow + ppnow;
	pprb = ppnow + dc_szx + 1;
	if (dc_szxy <= pprb) {
		pprb = pprb - dc_szxy;
	}
#define		Pnxt 	(d_pbasenxt + gy * dc_szx + gx)
#define		D_unow	d_u[pnow]
#define		D_vnow	d_v[pnow]
	uvv	= D_unow * D_vnow * D_vnow;
	d_u[Pnxt]					// Unxt(gy, gx) 
		= D_unow + DT * ((d_lsu[ppnow] + d_lsu[pprb]
						  - 4. * d_u[d_pbasenow + ppnow]) * dc_du_dxdx
						 + (-uvv + D_f * (1. - D_unow))
                        );
	d_v[Pnxt]					// Vnxt(gy, gx)
		= D_vnow + DT * ((d_lsv[ppnow] + d_lsv[pprb]
						  - 4. * d_v[d_pbasenow + ppnow]) * dc_dv_dxdx
						 + (uvv - (D_f + dc_k) * D_vnow)
						);

	return;
}
#endif	// CUDALSOPT


#if defined(CUDABLOPT)	// block optimized
__global__ void
cudagsblk(float *d_u, float *d_v, float *d_flny, float *d_lnx, int d_pswitch)
{
	int		by, bx;
	int		gy, gx;
	int		tx, ty, pbx, pby;
	float	uvv, lu, lv;
	int		d_pbasenow = (d_pswitch ? dc_pbase1 : dc_pbase0);
	int		d_pbasenxt = (d_pswitch ? dc_pbase0 : dc_pbase1);
	__shared__ float	ds_bu[CUDA_BLOCKSIZE][CUDA_BLOCKSIZE],
						ds_bv[CUDA_BLOCKSIZE][CUDA_BLOCKSIZE];

// this kernel doubled: index size be half of blocksize, same operation to
// left & right half at once corresponding to an index.
// reduce to half # of threads (only 2% faster...)

	by  = threadIdx.y;
	gy  = blockIdx.y * (blockDim.y - 2) + threadIdx.y - 1;
	bx = threadIdx.x;
//	gx = blockIdx.x * (blockDim.x - 2) + threadIdx.x - 1;
	gx = blockIdx.x * (CUDA_BLOCKSIZE - 2) + threadIdx.x - 1;
	if (dc_szy < gy || dc_szx < gx) {	// for block padding threads
		return;
	}

//........................................................................
// transfer (u, v) from global to shared memory
	tx = gx;
	ty = gy;
	if (tx < 0) {
		tx += dc_szx;
	} else if (dc_szx <= tx) {
		tx -= dc_szx;
	}
	if (ty < 0) {
		ty += dc_szy;
	} else if (dc_szy == ty) {
		ty -= dc_szy;
	}

#define	tnow	(d_pbasenow + ty * dc_szx + tx)
	ds_bu[by][bx] = d_u[tnow];
	ds_bv[by][bx] = d_v[tnow];
//........................................................................
	bx = threadIdx.x + CUDA_BLOCKSIZE / 2;
	tx = gx + CUDA_BLOCKSIZE / 2;
//	if (tx < 0) {
//		tx += dc_szx;
//	} else
	 if (dc_szx <= tx) {
		tx -= dc_szx;
	}
	ds_bu[by][bx] = d_u[tnow];
	ds_bv[by][bx] = d_v[tnow];

	__syncthreads();
//........................................................................
	bx = threadIdx.x;

// iteration:

// (pbx, pby) is iteration point in block,
// (gx, gy) is target point in global matrix.
	pbx = bx;
	pby = by;
	if (dc_edgetype == EDGE_NEUMANN) {
		if (gy == 0) {
			pby = pby + 1;	// should be 3
		} else if (gy == dc_szy - 1) {
			pby = pby - 1;
		}
		if (gx == 0) {
			pbx = pbx + 1;	// should be 3
		} else if (gx == dc_szx - 1) {
			pbx = pbx - 1;
		}
	}

#define		Pnxt 	(d_pbasenxt + gy * dc_szx + gx)
#define		D_f		(d_flny[gy] * d_lnx[gx])
	if (0 < bx && bx < CUDA_BLOCKSIZE - 1 &&
		0 < by && by < CUDA_BLOCKSIZE - 1 &&
			gx < dc_szx && gy < dc_szy) {
				// gx, gy should always be >0 if bx, by >=1.

		lu = (
			ds_bu[pby - 1][pbx]		// Unow(y - 1, x)
			+ ds_bu[pby][pbx - 1]	// Unow(y, x - 1)
			+ ds_bu[pby][pbx + 1]	// Unow(y, x + 1)
			+ ds_bu[pby + 1][pbx]	// Unow(y + 1, x)
			- 4. * ds_bu[pby][pbx]	// Unow(y, x)
		) * dc_du_dxdx;
		lv = (
			ds_bv[pby - 1][pbx]		// Vnow(y - 1, x)
			+ ds_bv[pby][pbx - 1]	// Vnow(y, x - 1)
			+ ds_bv[pby][pbx + 1]	// Vnow(y, x + 1)
			+ ds_bv[pby + 1][pbx]	// Vnow(y + 1, x)
			- 4. * ds_bv[pby][pbx]	// Vnow(y, x)
		) * dc_dv_dxdx;
		uvv	= ds_bu[pby][pbx] * ds_bv[pby][pbx] * ds_bv[pby][pbx];
		d_u[Pnxt]					// Unxt(gy, gx) 
			= ds_bu[pby][pbx] +
				DT * (lu - uvv + D_f * (1. - ds_bu[pby][pbx]));
		d_v[Pnxt]					// Vnxt(gy, gx)
			= ds_bv[pby][pbx] +
				DT * (lv + uvv - (D_f + dc_k) * ds_bv[pby][pbx]);
	}

//........................................................................
	bx += CUDA_BLOCKSIZE / 2;
//    gx = blockIdx.x * (blockDim.x - 2) + threadIdx.x - 1;
	gx += CUDA_BLOCKSIZE / 2;

	if (dc_szx <= gx) {	// for block padding threads
		return;
	}

	pbx = bx;
//	pby = by;
	if (dc_edgetype == EDGE_NEUMANN) {
//		if (gy == 0) {
//			pby = pby + 1;
//		} else if (gy == dc_szy - 1) {
//			pby = pby - 1;
//		}
		if (gx == 0) {
			pbx = pbx + 1;
		} else if (gx == dc_szx - 1) {
			pbx = pbx - 1;
		}
	}

//#define		Pnxt 	(d_pbasenxt + gy * dc_szx + gx)
//#define		D_f		(d_flny[gy] * d_lnx[gx])
	if (0 < bx && bx < CUDA_BLOCKSIZE - 1 &&
		0 < by && by < CUDA_BLOCKSIZE - 1 &&
			gx < dc_szx && gy < dc_szy) {
				// gx, gy should always be >0 if bx, by >=1.

		lu = (
			ds_bu[pby - 1][pbx]		// Unow(y - 1, x)
			+ ds_bu[pby][pbx - 1]	// Unow(y, x - 1)
			+ ds_bu[pby][pbx + 1]	// Unow(y, x + 1)
			+ ds_bu[pby + 1][pbx]	// Unow(y + 1, x)
			- 4. * ds_bu[pby][pbx]	// Unow(y, x)
		) * dc_du_dxdx;
		lv = (
			ds_bv[pby - 1][pbx]		// Vnow(y - 1, x)
			+ ds_bv[pby][pbx - 1]	// Vnow(y, x - 1)
			+ ds_bv[pby][pbx + 1]	// Vnow(y, x + 1)
			+ ds_bv[pby + 1][pbx]	// Vnow(y + 1, x)
			- 4. * ds_bv[pby][pbx]	// Vnow(y, x)
		) * dc_dv_dxdx;
		uvv	= ds_bu[pby][pbx] * ds_bv[pby][pbx] * ds_bv[pby][pbx];
		d_u[Pnxt]					// Unxt(gy, gx) 
			= ds_bu[pby][pbx] +
				DT * (lu - uvv + D_f * (1. - ds_bu[pby][pbx]));
		d_v[Pnxt]					// Vnxt(gy, gx)
			= ds_bv[pby][pbx] +
				DT * (lv + uvv - (D_f + dc_k) * ds_bv[pby][pbx]);
	}
//........................................................................

	return;
}
#endif	// optimize


#if defined(CUDAMAP)

void
cudamapdownload(void)
{
	cudamapbgr();
// download map
	if (hipMemcpy((img.ptr<cv::Vec3b>(0)), d_map, ZZMAP,
			hipMemcpyDeviceToHost) != hipSuccess
	   ) {
		exit(42);
	}
//for (int i = 0; i < 320*160*3; i++) {
//(img.ptr<unsigned char>(0, 0))[i] = 128;
//}
	return;
}


void
cudamapbgr(void)
{
	Float	umin, umax, vmin, vmax,
			udif, vdif, t;
	hipError_t		stat;

	dim3	mapdimBlock(CUDA_BLOCKSIZE, CUDA_BLOCKSIZE);
	dim3	mapdimGrid((szx + CUDA_BLOCKSIZE - 1) / (CUDA_BLOCKSIZE),
			        (szy + CUDA_BLOCKSIZE - 1) / (CUDA_BLOCKSIZE));

#if defined(CUDAFMM)
	cudafindminmax(&umin, &umax, &vmin, &vmax);
#else
	findminmax(&umin, &umax, &vmin, &vmax);
#endif
	udif = umax - umin;
	vdif = vmax - vmin;
    if (umax - umin < Epsilon) {
		udif = Epsilon;
	}
	if (vmax - vmin < Epsilon) {
		vdif = Epsilon;
	}

	cudamapk<<<mapdimGrid, mapdimBlock>>>(d_u, d_v, d_map, pswitch,
		udif, umin, vdif, vmin);
	if ((stat = hipGetLastError()) != hipSuccess) {
		fprintf(stderr, "(kernel) %s\n", hipGetErrorString(stat));
		exit(43);
	}

	return;
}


__global__ void
cudamapk(float *d_u, float *d_v, unsigned char *d_map, int d_pswitch,
	float d_udif, float d_umin, float d_vdif, float d_vmin)
{
	int		y = blockIdx.y * blockDim.y + threadIdx.y;
    int		x = blockIdx.x * blockDim.x + threadIdx.x;

	int		pnow;
	unsigned char		pr, pb, pg;

	pnow = (d_pswitch ? dc_pbase1 : dc_pbase0) + y * dc_szx + x;
#define		D_unow	d_u[pnow]
#define		D_vnow	d_v[pnow]
	pr = (unsigned char)((D_unow - d_umin) / d_udif * 255.);
	pb = (unsigned char)((D_vnow - d_vmin) / d_vdif * 255.);
	pg = (unsigned char)(255 - abs((int)pr - (int)pb));

	for (int yy = 0; yy < dc_mag; yy++) {
		for (int xx = 0; xx < dc_mag; xx++) {
			d_map[((y * dc_mag + yy) * (dc_szx * dc_mag)
					 + (x * dc_mag + xx)) * 3    ]
//				= (unsigned char)(x * 4);
				= pb;
			d_map[((y * dc_mag + yy) * (dc_szx * dc_mag)
					 + (x * dc_mag + xx)) * 3 + 1]
//				= (unsigned char)(x * 2);
				= pg;
			d_map[((y * dc_mag + yy) * (dc_szx * dc_mag)
					 + (x * dc_mag + xx)) * 3 + 2]
//				= (unsigned char)(x * 4);
				= pr;
		}
	}
	return;
}
#endif	// CUDAMAP


#if defined(CUDAFMM)
void
cudafindminmax(Float *umin, Float *umax, Float *vmin, Float *vmax)
{
// find (min, max) of each part #0..#(nsrch - 1) sized CUDA_FMMSIZE in d_u[],
//   and put into d_minmaxarr[#part * 2], d_minmaxarr[#part * 2 + 1]

//	if (CUDA_MAXBLOCKSIZE < szy) {
//		fprintf(stderr, "too large y size (cudafindminmax()).\n");
//		exit(45);
//	}
	int		nsrch = (szxy + CUDA_FMMSIZE - 1) / CUDA_FMMSIZE;
	dim3	mmdimBlock1(nsrch, 1);
	dim3	mmdimGrid1(1, 1);

	cudafindminmaxk<<<mmdimGrid1, mmdimBlock1>>>
		(d_u + (pswitch ? pbase1 : pbase0), CUDA_FMMSIZE, d_minmaxarr);

// find (min, max) in d_minmaxarr[0..(nsrch * 2 - 1)]
//   and put into minmaxres[0], minmaxres[1]
// note: here we don't have to use threads (#threads is 1),
//       but we can utilize same routine.
	dim3	mmdimBlock2(1, 1);
	dim3	mmdimGrid2(1, 1);
	cudafindminmaxk<<<mmdimGrid2, mmdimBlock2>>>
		(d_minmaxarr, nsrch * 2, d_minmaxres);

// same as above, on d_v[]
//	dim3	mmdimBlock1(CUDA_BLOCKSIZE, CUDA_BLOCKSIZE);
//	dim3	mmdimGrid1((szy + CUDA_BLOCKSIZE - 1) / (CUDA_BLOCKSIZE), 1);
	cudafindminmaxk<<<mmdimGrid1, mmdimBlock1>>>
		(d_v + (pswitch ? pbase1 : pbase0), CUDA_FMMSIZE, d_minmaxarr);

//	dim3	mmdimBlock2(1, 1);
//	dim3	mmdimGrid2(1, 1);
	cudafindminmaxk<<<mmdimGrid2, mmdimBlock2>>>
		(d_minmaxarr, nsrch * 2, d_minmaxres + 2);

// now d_minmaxres[0..3] is {umin, umax, vmin, vmax}.
	if (hipMemcpy(minmaxres, d_minmaxres, sizeof(float) * 4,
			hipMemcpyDeviceToHost) != hipSuccess) {
		exit(42);
	}
	*umin = minmaxres[0];
	*umax = minmaxres[1];
	*vmin = minmaxres[2];
	*vmax = minmaxres[3];
	
	return;
}


__global__ void
cudafindminmaxk(float source[], int srchsize, float result[])
{
	int		part = threadIdx.x;
	float	min = 99., max = -99.;

	for (int i = part * srchsize; i < (part + 1) * srchsize; i++) {
		if (dc_szxy <= i) {	// protect exceeding u[]/v[] when
			break;			// szxy is not multiply of CUDA_FMMSIZE.
		}					// when summarize, srchsize never exceeds szxy.
		if (source[i] < min) {
			min = source[i];
		}
		if (max < source[i]) {
			max = source[i];
		}
	}
	result[part * 2    ] = min;
	result[part * 2 + 1] = max;

	return;
}
#endif	// CUDAFMM

#endif	// CUDA
